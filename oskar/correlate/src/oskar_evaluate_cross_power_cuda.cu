#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014-2015, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "correlate/oskar_evaluate_cross_power_cuda.h"
#include "correlate/private_correlate_functions_inline.h"

/* Kernels. ================================================================ */

extern __shared__ float2  smem_2f[];
extern __shared__ double2 smem_2d[];
extern __shared__ float4c  smem_4f[];
extern __shared__ double4c smem_4d[];

/* Single precision. */
__global__
void oskar_evaluate_cross_power_cudak_f(const int num_sources,
        const int num_stations, const float4c* restrict jones,
        float4c* restrict out, const float norm)
{
    float4c val1, val2, *p, q;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate cross-power product at the source. */
    p = &smem_4f[threadIdx.x];
    oskar_clear_complex_matrix_f(&val1);
    for (int SP = 0; SP < num_stations; ++SP)
    {
        /* Load data for first station into shared memory. */
        OSKAR_LOAD_MATRIX(smem_4f[threadIdx.x], jones, SP * num_sources + i);
        oskar_clear_complex_matrix_f(&val2);

        /* Cross-correlate. */
        for (int SQ = SP + 1; SQ < num_stations; ++SQ)
        {
            /* Load data for second station into registers. */
            OSKAR_LOAD_MATRIX(q, jones, SQ * num_sources + i);

            /* Multiply-add: val += p * conj(q). */
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.a, p->a, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.a, p->b, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.b, p->a, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.b, p->b, q.d);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.c, p->c, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.c, p->d, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.d, p->c, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2.d, p->d, q.d);
        }

        /* Accumulate partial sum (try to preserve numerical precision). */
        val1.a.x += val2.a.x;
        val1.a.y += val2.a.y;
        val1.b.x += val2.b.x;
        val1.b.y += val2.b.y;
        val1.c.x += val2.c.x;
        val1.c.y += val2.c.y;
        val1.d.x += val2.d.x;
        val1.d.y += val2.d.y;
    }

    /* Calculate average by dividing by number of baselines. */
    val1.a.x *= norm;
    val1.a.y *= norm;
    val1.b.x *= norm;
    val1.b.y *= norm;
    val1.c.x *= norm;
    val1.c.y *= norm;
    val1.d.x *= norm;
    val1.d.y *= norm;

    /* Store result. */
    out[i] = val1;
}

__global__
void oskar_evaluate_cross_power_scalar_cudak_f(
        const int num_sources, const int num_stations,
        const float2* restrict jones, float2* restrict out,
        const float norm)
{
    float2 val1, val2, q;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate cross-power product at the source. */
    val1.x = 0.0f;
    val1.y = 0.0f;
    for (int SP = 0; SP < num_stations; ++SP)
    {
        /* Load data for first station into shared memory. */
        smem_2f[threadIdx.x] = jones[SP * num_sources + i];
        val2.x = 0.0f;
        val2.y = 0.0f;

        /* Cross-correlate. */
        for (int SQ = SP + 1; SQ < num_stations; ++SQ)
        {
            /* Load data for second station into registers. */
            q = jones[SQ * num_sources + i];

            /* Multiply-add: val += p * conj(q). */
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val2, smem_2f[threadIdx.x], q);
        }

        /* Accumulate partial sum (try to preserve numerical precision). */
        val1.x += val2.x;
        val1.y += val2.y;
    }

    /* Calculate average by dividing by number of baselines. */
    val1.x *= norm;
    val1.y *= norm;

    /* Store result. */
    out[i] = val1;
}

/* Double precision. */
__global__
void oskar_evaluate_cross_power_scalar_cudak_d(
        const int num_sources, const int num_stations,
        const double2* restrict jones, double2* restrict out,
        const double norm)
{
    double2 val1, val2, q;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate cross-power product at the source. */
    val1.x = 0.0;
    val1.y = 0.0;
    for (int SP = 0; SP < num_stations; ++SP)
    {
        /* Load data for first station into shared memory. */
        smem_2d[threadIdx.x] = jones[SP * num_sources + i];
        val2.x = 0.0;
        val2.y = 0.0;

        /* Cross-correlate. */
        for (int SQ = SP + 1; SQ < num_stations; ++SQ)
        {
            /* Load data for second station into registers. */
            q = jones[SQ * num_sources + i];

            /* Multiply-add: val += p * conj(q). */
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val1, smem_2d[threadIdx.x], q);
        }

        /* Accumulate partial sum (try to preserve numerical precision). */
        val1.x += val2.x;
        val1.y += val2.y;
    }

    /* Calculate average by dividing by number of baselines. */
    val1.x *= norm;
    val1.y *= norm;

    /* Store result. */
    out[i] = val1;
}

__global__
void oskar_evaluate_cross_power_cudak_d(const int num_sources,
        const int num_stations, const double4c* restrict jones,
        double4c* restrict out, const double norm)
{
    double4c val, *p, q;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate cross-power product at the source. */
    p = &smem_4d[threadIdx.x];
    oskar_clear_complex_matrix_d(&val);
    for (int SP = 0; SP < num_stations; ++SP)
    {
        /* Load data for first station into shared memory. */
        OSKAR_LOAD_MATRIX(smem_4d[threadIdx.x], jones, SP * num_sources + i);

        /* Cross-correlate. */
        for (int SQ = SP + 1; SQ < num_stations; ++SQ)
        {
            /* Load data for second station into registers. */
            OSKAR_LOAD_MATRIX(q, jones, SQ * num_sources + i);

            /* Multiply-add: val += p * conj(q). */
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.a, p->a, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.a, p->b, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.b, p->a, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.b, p->b, q.d);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.c, p->c, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.c, p->d, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.d, p->c, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.d, p->d, q.d);
        }
    }

    /* Calculate average by dividing by number of baselines. */
    val.a.x *= norm;
    val.a.y *= norm;
    val.b.x *= norm;
    val.b.y *= norm;
    val.c.x *= norm;
    val.c.y *= norm;
    val.d.x *= norm;
    val.d.y *= norm;

    /* Store result. */
    out[i] = val;
}

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_evaluate_cross_power_cuda_f(int num_sources,
        int num_stations, const float4c* d_jones, float4c* d_out)
{
    int num_blocks, num_threads = 128;
    size_t shared_mem = num_threads * sizeof(float4c);
    float norm = 2.0f / (num_stations * (num_stations - 1));
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_cross_power_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem) (num_sources,
            num_stations, d_jones, d_out, norm);
}

void oskar_evaluate_cross_power_scalar_cuda_f(int num_sources,
        int num_stations, const float2* d_jones, float2* d_out)
{
    int num_blocks, num_threads = 128;
    size_t shared_mem = num_threads * sizeof(float2);
    float norm = 2.0f / (num_stations * (num_stations - 1));
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_cross_power_scalar_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem) (num_sources,
            num_stations, d_jones, d_out, norm);
}

/* Double precision. */
void oskar_evaluate_cross_power_cuda_d(int num_sources,
        int num_stations, const double4c* d_jones, double4c* d_out)
{
    int num_blocks, num_threads = 64;
    size_t shared_mem = num_threads * sizeof(double4c);
    double norm = 2.0 / (num_stations * (num_stations - 1));
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_cross_power_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem) (num_sources,
            num_stations, d_jones, d_out, norm);
}

void oskar_evaluate_cross_power_scalar_cuda_d(int num_sources,
        int num_stations, const double2* d_jones, double2* d_out)
{
    int num_blocks, num_threads = 128;
    size_t shared_mem = num_threads * sizeof(double2);
    double norm = 2.0 / (num_stations * (num_stations - 1));
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_cross_power_scalar_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem) (num_sources,
            num_stations, d_jones, d_out, norm);
}

#ifdef __cplusplus
}
#endif
