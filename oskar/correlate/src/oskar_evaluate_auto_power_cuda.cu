#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "correlate/oskar_evaluate_auto_power_cuda.h"
#include "correlate/private_correlate_functions_inline.h"

/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_evaluate_auto_power_cudak_f(const int num_sources,
        const float4c* restrict jones, float4c* restrict out)
{
    float4c val1, val2;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate auto-power product at the source. */
    OSKAR_LOAD_MATRIX(val1, jones, i);
    val2 = val1;
    oskar_multiply_complex_matrix_conjugate_transpose_in_place_f(&val1, &val2);

    /* Store result. */
    out[i] = val1;
}

__global__
void oskar_evaluate_auto_power_scalar_cudak_f(const int num_sources,
        const float2* restrict jones, float2* restrict out)
{
    float2 val1, val2;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate auto-power product at the source. */
    val1 = jones[i];
    val2 = val1;
    oskar_multiply_complex_conjugate_in_place_f(&val1, &val2);

    /* Store result. */
    out[i] = val1;
}

/* Double precision. */
__global__
void oskar_evaluate_auto_power_cudak_d(const int num_sources,
        const double4c* restrict jones, double4c* restrict out)
{
    double4c val1, val2;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate auto-power product at the source. */
    OSKAR_LOAD_MATRIX(val1, jones, i);
    val2 = val1;
    oskar_multiply_complex_matrix_conjugate_transpose_in_place_d(&val1, &val2);

    /* Store result. */
    out[i] = val1;
}

__global__
void oskar_evaluate_auto_power_scalar_cudak_d(const int num_sources,
        const double2* restrict jones, double2* restrict out)
{
    double2 val1, val2;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate auto-power product at the source. */
    val1 = jones[i];
    val2 = val1;
    oskar_multiply_complex_conjugate_in_place_d(&val1, &val2);

    /* Store result. */
    out[i] = val1;
}

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_evaluate_auto_power_cuda_f(int num_sources,
        const float4c* d_jones, float4c* d_out)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_auto_power_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_sources, d_jones, d_out);
}

void oskar_evaluate_auto_power_scalar_cuda_f(int num_sources,
        const float2* d_jones, float2* d_out)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_auto_power_scalar_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_sources, d_jones, d_out);
}

/* Double precision. */
void oskar_evaluate_auto_power_cuda_d(int num_sources,
        const double4c* d_jones, double4c* d_out)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_auto_power_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_sources, d_jones, d_out);
}

void oskar_evaluate_auto_power_scalar_cuda_d(int num_sources,
        const double2* d_jones, double2* d_out)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_auto_power_scalar_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_sources, d_jones, d_out);
}

#ifdef __cplusplus
}
#endif
