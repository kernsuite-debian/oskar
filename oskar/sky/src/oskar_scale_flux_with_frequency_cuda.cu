#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011-2014, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "sky/oskar_scale_flux_with_frequency_cuda.h"
#include "sky/oskar_scale_flux_with_frequency_inline.h"

/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_scale_flux_with_frequency_cudak_f(const int num_sources,
        const float frequency, float* restrict I, float* restrict Q,
        float* restrict U, float* restrict V,
        float* restrict ref_freq, const float* restrict sp_index,
        const float* restrict rm)
{
    /* Get source index and check bounds. */
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    oskar_scale_flux_with_frequency_inline_f(frequency,
            &I[i], &Q[i], &U[i], &V[i], &ref_freq[i], sp_index[i], rm[i]);
}

/* Double precision. */
__global__
void oskar_scale_flux_with_frequency_cudak_d(const int num_sources,
        const double frequency, double* restrict I, double* restrict Q,
        double* restrict U, double* restrict V,
        double* restrict ref_freq, const double* restrict sp_index,
        const double* restrict rm)
{
    /* Get source index and check bounds. */
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    oskar_scale_flux_with_frequency_inline_d(frequency,
            &I[i], &Q[i], &U[i], &V[i], &ref_freq[i], sp_index[i], rm[i]);
}

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_scale_flux_with_frequency_cuda_f(int num_sources,
        float frequency, float* d_I, float* d_Q, float* d_U, float* d_V,
        float* d_ref_freq, const float* d_sp_index, const float* d_rm)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_scale_flux_with_frequency_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_sources, frequency,
            d_I, d_Q, d_U, d_V, d_ref_freq, d_sp_index, d_rm);
}

/* Double precision. */
void oskar_scale_flux_with_frequency_cuda_d(int num_sources,
        double frequency, double* d_I, double* d_Q, double* d_U, double* d_V,
        double* d_ref_freq, const double* d_sp_index, const double* d_rm)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_scale_flux_with_frequency_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_sources, frequency,
            d_I, d_Q, d_U, d_V, d_ref_freq, d_sp_index, d_rm);
}

#ifdef __cplusplus
}
#endif
