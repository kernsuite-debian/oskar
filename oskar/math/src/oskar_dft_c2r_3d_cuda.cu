#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "math/oskar_dft_c2r_3d_cuda.h"


/* Kernels. ================================================================ */

/* Shared memory pointers used by the kernels. */
extern __shared__ float2 smem_f[];
extern __shared__ double2 smem_d[];

/* Single precision. */
__global__
void oskar_dft_c2r_3d_cudak_f(int n_in,
        const float wavenumber,
        const float* __restrict__ x_in,
        const float* __restrict__ y_in,
        const float* __restrict__ z_in,
        const float2* __restrict__ data_in,
        const float* __restrict__ weight_in,
        const int n_out,
        const float* __restrict__ x_out,
        const float* __restrict__ y_out,
        const float* __restrict__ z_out,
        const int max_in_chunk,
        float* __restrict__ output)
{
    // Get the output position (pixel) ID that this thread is working on.
    const int i_out = blockDim.x * blockIdx.x + threadIdx.x;
    float out = 0.0f; // Clear output value.

    // Initialise shared memory caches.
    float2* cd = smem_f; // Cached input data.
    float2* cp = cd + max_in_chunk; // Cached input x,y positions.
    float* cz = (float*)(cp + max_in_chunk); // Cached input z positions.

    // Get the output position.
    // (NB. Cannot exit on index condition, as all threads are needed later.)
    float xp_out = 0.0f, yp_out = 0.0f, zp_out = 0.0f;
    if (i_out < n_out)
    {
        xp_out = wavenumber * x_out[i_out];
        yp_out = wavenumber * y_out[i_out];
        zp_out = wavenumber * z_out[i_out];
    }

    // Cache a chunk of input data and positions into shared memory.
    for (int start = 0; start < n_in; start += max_in_chunk)
    {
        int chunk_size = n_in - start;
        if (chunk_size > max_in_chunk)
            chunk_size = max_in_chunk;

        // There are blockDim.x threads available - need to copy
        // chunk_size pieces of data from global memory.
        for (int t = threadIdx.x; t < chunk_size; t += blockDim.x)
        {
            const int g = start + t; // Global input index.
            cd[t] = data_in[g];
            cp[t].x = x_in[g];
            cp[t].y = y_in[g];
            cz[t] = z_in[g];
            cd[t].x *= weight_in[g];
            cd[t].y *= weight_in[g];
        }

        // Must synchronise before computing partial output for these inputs.
        __syncthreads();

        // Loop over input block.
        for (int i = 0; i < chunk_size; ++i)
        {
            // Calculate the complex DFT weight.
            float2 weight, d = cd[i];
            float a = xp_out * cp[i].x + yp_out * cp[i].y + zp_out * cz[i];
            sincosf(-a, &weight.y, &weight.x);

            // Perform complex multiply-accumulate.
            // Output is real, so only evaluate the real part.
            out += d.x * weight.x; // RE*RE
            out -= d.y * weight.y; // IM*IM
        }

        // Must synchronise again before loading in a new input block.
        __syncthreads();
    }

    // Copy result into global memory.
    if (i_out < n_out)
        output[i_out] = out;
}

/* Double precision. */
__global__
void oskar_dft_c2r_3d_cudak_d(int n_in,
        const double wavenumber,
        const double* __restrict__ x_in,
        const double* __restrict__ y_in,
        const double* __restrict__ z_in,
        const double2* __restrict__ data_in,
        const double* __restrict__ weight_in,
        const int n_out,
        const double* __restrict__ x_out,
        const double* __restrict__ y_out,
        const double* __restrict__ z_out,
        const int max_in_chunk,
        double* __restrict__ output)
{
    // Get the output position (pixel) ID that this thread is working on.
    const int i_out = blockDim.x * blockIdx.x + threadIdx.x;
    double out = 0.0; // Clear output value.

    // Initialise shared memory caches.
    double2* cd = smem_d; // Cached input data.
    double2* cp = cd + max_in_chunk; // Cached input x,y positions.
    double* cz = (double*)(cp + max_in_chunk); // Cached input z positions.

    // Get the output position.
    // (NB. Cannot exit on index condition, as all threads are needed later.)
    double xp_out = 0.0, yp_out = 0.0, zp_out = 0.0;
    if (i_out < n_out)
    {
        xp_out = wavenumber * x_out[i_out];
        yp_out = wavenumber * y_out[i_out];
        zp_out = wavenumber * z_out[i_out];
    }

    // Cache a chunk of input data and positions into shared memory.
    for (int start = 0; start < n_in; start += max_in_chunk)
    {
        int chunk_size = n_in - start;
        if (chunk_size > max_in_chunk)
            chunk_size = max_in_chunk;

        // There are blockDim.x threads available - need to copy
        // chunk_size pieces of data from global memory.
        for (int t = threadIdx.x; t < chunk_size; t += blockDim.x)
        {
            const int g = start + t; // Global input index.
            cd[t] = data_in[g];
            cp[t].x = x_in[g];
            cp[t].y = y_in[g];
            cz[t] = z_in[g];
            cd[t].x *= weight_in[g];
            cd[t].y *= weight_in[g];
        }

        // Must synchronise before computing partial output for these inputs.
        __syncthreads();

        // Loop over input block.
        for (int i = 0; i < chunk_size; ++i)
        {
            // Calculate the complex DFT weight.
            double2 weight, d = cd[i];
            double a = xp_out * cp[i].x + yp_out * cp[i].y + zp_out * cz[i];
            sincos(-a, &weight.y, &weight.x);

            // Perform complex multiply-accumulate.
            // Output is real, so only evaluate the real part.
            out += d.x * weight.x; // RE*RE
            out -= d.y * weight.y; // IM*IM
        }

        // Must synchronise again before loading in a new input block.
        __syncthreads();
    }

    // Copy result into global memory.
    if (i_out < n_out)
        output[i_out] = out;
}

#ifdef __cplusplus
extern "C" {
#endif

/* Utility functions. */
static int oskar_int_round_to_nearest_multiple(int num_to_round, int multiple)
{
   return (num_to_round + multiple - 1) / multiple * multiple;
}

static int oskar_int_range_clamp(int value, int minimum, int maximum)
{
   if (value < minimum)
       return minimum;
   if (value > maximum)
       return maximum;
   return value;
}


/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_dft_c2r_3d_cuda_f(int num_in, float wavenumber, const float* x_in,
        const float* y_in, const float* z_in, const float2* data_in,
        const float* weight_in, int num_out, const float* x_out,
        const float* y_out, const float* z_out, float* output)
{
    const int threads = 384;     /* Should be multiple of 32. */
    const int max_in_size = 800; /* Should be multiple of 16. */
    int out_size, max_out_size, blocks, shared_mem_size, start;

    /* Initialise. */
    shared_mem_size = 5 * max_in_size * sizeof(float);

    /* Compute the maximum manageable output chunk size. */
    max_out_size = 65536 * 8192; /* Product of max output and input sizes. */
    max_out_size /= num_in;
    max_out_size = oskar_int_round_to_nearest_multiple(max_out_size, threads);
    max_out_size = oskar_int_range_clamp(max_out_size,
            2 * threads, 160 * threads);

    /* Loop over output chunks. */
    for (start = 0; start < num_out; start += max_out_size)
    {
        out_size = num_out - start;
        if (out_size > max_out_size) out_size = max_out_size;

        /* Invoke kernel to compute the (partial) DFT on the device. */
        blocks = (out_size + threads - 1) / threads;
        oskar_dft_c2r_3d_cudak_f
        OSKAR_CUDAK_CONF(blocks, threads, shared_mem_size) (num_in, wavenumber,
                x_in, y_in, z_in, data_in, weight_in, out_size, x_out + start,
                y_out + start, z_out + start, max_in_size, output + start);
    }
}

/* Double precision. */
void oskar_dft_c2r_3d_cuda_d(int num_in, double wavenumber, const double* x_in,
        const double* y_in, const double* z_in, const double2* data_in,
        const double* weight_in, int num_out, const double* x_out,
        const double* y_out, const double* z_out, double* output)
{
    const int threads = 384;     /* Should be multiple of 32. */
    const int max_in_size = 384; /* Should be multiple of 16. */
    int out_size, max_out_size, blocks, shared_mem_size, start;

    /* Initialise. */
    shared_mem_size = 5 * max_in_size * sizeof(double);

    /* Compute the maximum manageable output chunk size. */
    max_out_size = 32768 * 8192; /* Product of max output and input sizes. */
    max_out_size /= num_in;
    max_out_size = oskar_int_round_to_nearest_multiple(max_out_size, threads);
    max_out_size = oskar_int_range_clamp(max_out_size,
            2 * threads, 80 * threads);

    /* Loop over output chunks. */
    for (start = 0; start < num_out; start += max_out_size)
    {
        out_size = num_out - start;
        if (out_size > max_out_size) out_size = max_out_size;

        /* Invoke kernel to compute the (partial) DFT on the device. */
        blocks = (out_size + threads - 1) / threads;
        oskar_dft_c2r_3d_cudak_d
        OSKAR_CUDAK_CONF(blocks, threads, shared_mem_size) (num_in, wavenumber,
                x_in, y_in, z_in, data_in, weight_in, out_size, x_out + start,
                y_out + start, z_out + start, max_in_size, output + start);
    }
}

#ifdef __cplusplus
}
#endif
