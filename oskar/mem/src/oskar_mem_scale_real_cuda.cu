#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "mem/oskar_mem_scale_real_cuda.h"

/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_mem_scale_real_cudak_f(int num, float value, float* a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num)
        a[i] *= value;
}

/* Double precision. */
__global__
void oskar_mem_scale_real_cudak_d(int num, double value, double* a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num)
        a[i] *= value;
}

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_mem_scale_real_cuda_f(int num, float value, float* a)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_mem_scale_real_cudak_f OSKAR_CUDAK_CONF(num_blocks, num_threads)
            (num, value, a);
}

/* Double precision. */
void oskar_mem_scale_real_cuda_d(int num, double value, double* a)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_mem_scale_real_cudak_d OSKAR_CUDAK_CONF(num_blocks, num_threads)
            (num, value, a);
}
