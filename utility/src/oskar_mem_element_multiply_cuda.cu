#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <private_mem.h>
#include <oskar_mem.h>

#include <oskar_multiply_inline.h>
#include <oskar_cuda_check_error.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Single precision. */
__global__
void oskar_element_multiply_cudak_rr_r_f(const int n, const float* a,
        const float* b, float* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] * b[i];
}

__global__
void oskar_element_multiply_cudak_cc_c_f(const int n, const float2* a,
        const float2* b, float2* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        float2 ac, bc, cc;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_f(&cc, &ac, &bc);
        c[i] = cc;
    }
}

__global__
void oskar_element_multiply_cudak_cc_m_f(const int n, const float2* a,
        const float2* b, float4c* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        float2 ac, bc, cc;
        float4c m;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_f(&cc, &ac, &bc);

        /* Store result in a matrix. */
        m.a = cc;
        m.b.x = 0.0f;
        m.b.y = 0.0f;
        m.c.x = 0.0f;
        m.c.y = 0.0f;
        m.d = cc;
        c[i] = m;
    }
}

__global__
void oskar_element_multiply_cudak_cm_m_f(const int n, const float2* a,
        const float4c* b, float4c* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        float2 ac;
        float4c bc;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_matrix_complex_scalar_in_place_f(&bc, &ac);
        c[i] = bc;
    }
}

__global__
void oskar_element_multiply_cudak_mm_m_f(const int n, const float4c* a,
        const float4c* b, float4c* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        float4c ac, bc;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_matrix_in_place_f(&ac, &bc);
        c[i] = ac;
    }
}

void oskar_mem_element_multiply_cuda_rr_r_f(int num, float* d_c,
        const float* d_a, const float* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_rr_r_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_cc_c_f(int num, float2* d_c,
        const float2* d_a, const float2* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_cc_c_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_cc_m_f(int num, float4c* d_c,
        const float2* d_a, const float2* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_cc_m_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_cm_m_f(int num, float4c* d_c,
        const float2* d_a, const float4c* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_cm_m_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_mm_m_f(int num, float4c* d_c,
        const float4c* d_a, const float4c* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_mm_m_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}


/* Double precision. */
__global__
void oskar_element_multiply_cudak_rr_r_d(const int n, const double* a,
        const double* b, double* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] * b[i];
}

__global__
void oskar_element_multiply_cudak_cc_c_d(const int n, const double2* a,
        const double2* b, double2* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        double2 ac, bc, cc;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_d(&cc, &ac, &bc);
        c[i] = cc;
    }
}

__global__
void oskar_element_multiply_cudak_cc_m_d(const int n, const double2* a,
        const double2* b, double4c* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        double2 ac, bc, cc;
        double4c m;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_d(&cc, &ac, &bc);

        /* Store result in a matrix. */
        m.a = cc;
        m.b.x = 0.0;
        m.b.y = 0.0;
        m.c.x = 0.0;
        m.c.y = 0.0;
        m.d = cc;
        c[i] = m;
    }
}

__global__
void oskar_element_multiply_cudak_cm_m_d(const int n, const double2* a,
        const double4c* b, double4c* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        double2 ac;
        double4c bc;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_matrix_complex_scalar_in_place_d(&bc, &ac);
        c[i] = bc;
    }
}

__global__
void oskar_element_multiply_cudak_mm_m_d(const int n, const double4c* a,
        const double4c* b, double4c* c)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        double4c ac, bc;
        ac = a[i];
        bc = b[i];
        oskar_multiply_complex_matrix_in_place_d(&ac, &bc);
        c[i] = ac;
    }
}

void oskar_mem_element_multiply_cuda_rr_r_d(int num, double* d_c,
        const double* d_a, const double* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_rr_r_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_cc_c_d(int num, double2* d_c,
        const double2* d_a, const double2* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_cc_c_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_cc_m_d(int num, double4c* d_c,
        const double2* d_a, const double2* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_cc_m_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_cm_m_d(int num, double4c* d_c,
        const double2* d_a, const double4c* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_cm_m_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}

void oskar_mem_element_multiply_cuda_mm_m_d(int num, double4c* d_c,
        const double4c* d_a, const double4c* d_b)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num + num_threads - 1) / num_threads;
    oskar_element_multiply_cudak_mm_m_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num, d_a, d_b, d_c);
}


static void oskar_mem_element_multiply_select_cuda(oskar_Mem* c,
        const oskar_Mem* a, const oskar_Mem* b, size_t num, int* status)
{
    int error = OSKAR_ERR_TYPE_MISMATCH; /* Set to type mismatch by default. */

    /* Cast num to int. Yes, this is horrible, but if num is really that big,
     * we'll exceed the maximum CUDA grid dimensions anyway. */
    int n;
    n = (int) num;

    /* Check if safe to proceed. */
    if (*status) return;

    /* Multiply the elements. */
    if (a->type == OSKAR_DOUBLE)
    {
        if (b->type == OSKAR_DOUBLE)
        {
            if (c->type == OSKAR_DOUBLE)
            {
                /* Real, real to real. */
                error = 0;
                oskar_mem_element_multiply_cuda_rr_r_d(n, (double*)c->data,
                        (const double*)a->data, (const double*)b->data);
            }
        }
    }
    else if (a->type == OSKAR_DOUBLE_COMPLEX)
    {
        if (b->type == OSKAR_DOUBLE_COMPLEX)
        {
            if (c->type == OSKAR_DOUBLE_COMPLEX)
            {
                /* Complex scalar, complex scalar to complex scalar. */
                error = 0;
                oskar_mem_element_multiply_cuda_cc_c_d(n, (double2*)c->data,
                        (const double2*)a->data, (const double2*)b->data);
            }
            else if (c->type == OSKAR_DOUBLE_COMPLEX_MATRIX)
            {
                /* Complex scalar, complex scalar to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_cc_m_d(n, (double4c*)c->data,
                        (const double2*)a->data, (const double2*)b->data);
            }
        }
        else if (b->type == OSKAR_DOUBLE_COMPLEX_MATRIX)
        {
            if (c->type == OSKAR_DOUBLE_COMPLEX_MATRIX)
            {
                /* Complex scalar, complex matrix to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_cm_m_d(n, (double4c*)c->data,
                        (const double2*)a->data, (const double4c*)b->data);
            }
        }
    }
    else if (a->type == OSKAR_DOUBLE_COMPLEX_MATRIX)
    {
        if (b->type == OSKAR_DOUBLE_COMPLEX)
        {
            if (c->type == OSKAR_DOUBLE_COMPLEX_MATRIX)
            {
                /* Complex matrix, complex scalar to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_cm_m_d(n, (double4c*)c->data,
                        (const double2*)b->data, (const double4c*)a->data);
            }
        }
        else if (b->type == OSKAR_DOUBLE_COMPLEX_MATRIX)
        {
            if (c->type == OSKAR_DOUBLE_COMPLEX_MATRIX)
            {
                /* Complex matrix, complex matrix to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_mm_m_d(n, (double4c*)c->data,
                        (const double4c*)a->data, (const double4c*)b->data);
            }
        }
    }
    else if (a->type == OSKAR_SINGLE)
    {
        if (b->type == OSKAR_SINGLE)
        {
            if (c->type == OSKAR_SINGLE)
            {
                /* Real, real to real. */
                error = 0;
                oskar_mem_element_multiply_cuda_rr_r_f(n, (float*)c->data,
                        (const float*)a->data, (const float*)b->data);
            }
        }
    }
    else if (a->type == OSKAR_SINGLE_COMPLEX)
    {
        if (b->type == OSKAR_SINGLE_COMPLEX)
        {
            if (c->type == OSKAR_SINGLE_COMPLEX)
            {
                /* Complex scalar, complex scalar to complex scalar. */
                error = 0;
                oskar_mem_element_multiply_cuda_cc_c_f(n, (float2*)c->data,
                        (const float2*)a->data, (const float2*)b->data);
            }
            else if (c->type == OSKAR_SINGLE_COMPLEX_MATRIX)
            {
                /* Complex scalar, complex scalar to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_cc_m_f(n, (float4c*)c->data,
                        (const float2*)a->data, (const float2*)b->data);
            }
        }
        else if (b->type == OSKAR_SINGLE_COMPLEX_MATRIX)
        {
            if (c->type == OSKAR_SINGLE_COMPLEX_MATRIX)
            {
                /* Complex scalar, complex matrix to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_cm_m_f(n, (float4c*)c->data,
                        (const float2*)a->data, (const float4c*)b->data);
            }
        }
    }
    else if (a->type == OSKAR_SINGLE_COMPLEX_MATRIX)
    {
        if (b->type == OSKAR_SINGLE_COMPLEX)
        {
            if (c->type == OSKAR_SINGLE_COMPLEX_MATRIX)
            {
                /* Complex matrix, complex scalar to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_cm_m_f(n, (float4c*)c->data,
                        (const float2*)b->data, (const float4c*)a->data);
            }
        }
        else if (b->type == OSKAR_SINGLE_COMPLEX_MATRIX)
        {
            if (c->type == OSKAR_SINGLE_COMPLEX_MATRIX)
            {
                /* Complex matrix, complex matrix to complex matrix. */
                error = 0;
                oskar_mem_element_multiply_cuda_mm_m_f(n, (float4c*)c->data,
                        (const float4c*)a->data, (const float4c*)b->data);
            }
        }
    }

    /* Check for type mismatch and CUDA error. */
    if (error) *status = error;
    oskar_cuda_check_error(status);
}


void oskar_mem_element_multiply_cuda(oskar_Mem* C, const oskar_Mem* A,
        const oskar_Mem* B, size_t num, int* status)
{
    oskar_Mem *At = 0, *Bt = 0;
    const oskar_Mem *Ap, *Bp;

    /* Check if safe to proceed. */
    if (*status) return;

    /* Set default pointer values. */
    Ap = A;
    Bp = B;

    /* Check that the output array is in GPU memory. */
    if (oskar_mem_location(C) != OSKAR_GPU)
    {
        *status = OSKAR_ERR_BAD_LOCATION;
        return;
    }

    /* Check memory is allocated. */
    if (!oskar_mem_allocated(A) || !oskar_mem_allocated(B) ||
            !oskar_mem_allocated(C))
    {
        *status = OSKAR_ERR_MEMORY_NOT_ALLOCATED;
        return;
    }

    /* Set the number of elements to multiply. */
    if (num <= 0) num = oskar_mem_length(A);

    /* Check that there are enough elements. */
    if (oskar_mem_length(B) < num || oskar_mem_length(C) < num)
    {
        *status = OSKAR_ERR_DIMENSION_MISMATCH;
        return;
    }

    /* Copy input data to temporary GPU memory if required. */
    if (oskar_mem_location(A) != OSKAR_GPU)
    {
        At = oskar_mem_create_copy(A, OSKAR_GPU, status);
        Ap = At;
    }
    if (oskar_mem_location(B) != OSKAR_GPU)
    {
        Bt = oskar_mem_create_copy(B, OSKAR_GPU, status);
        Bp = Bt;
    }

    /* Do the multiplication using CUDA. */
    oskar_mem_element_multiply_select_cuda(C, Ap, Bp, num, status);

    /* Free temporary arrays if they exist. */
    oskar_mem_free(At, status);
    oskar_mem_free(Bt, status);
}

#ifdef __cplusplus
}
#endif
