#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_convert_enu_directions_to_relative_directions_cuda.h>
#include <oskar_convert_enu_directions_to_relative_directions_inline.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

void oskar_convert_enu_directions_to_relative_directions_cuda_f(
        float* l, float* m, float* n, int num_points, const float* x,
        const float* y, const float* z, float ha0, float dec0, float lat)
{
    float sin_ha0, cos_ha0, sin_dec0, cos_dec0, sin_lat, cos_lat;
    int num_blocks, num_threads = 256;

    /* Set up thread blocks. */
    num_blocks = (num_points + num_threads - 1) / num_threads;

    /* Compute sines and cosines of Euler angles and call kernel to perform
     * the transformation. */
    sin_ha0  = (float) sin(ha0);
    cos_ha0  = (float) cos(ha0);
    sin_dec0 = (float) sin(dec0);
    cos_dec0 = (float) cos(dec0);
    sin_lat  = (float) sin(lat);
    cos_lat  = (float) cos(lat);
    oskar_convert_enu_directions_to_relative_directions_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (l, m, n, num_points, x, y, z,
            cos_ha0, sin_ha0, cos_dec0, sin_dec0, cos_lat, sin_lat);
}

void oskar_convert_enu_directions_to_relative_directions_cuda_d(
        double* l, double* m, double* n, int num_points, const double* x,
        const double* y, const double* z, double ha0, double dec0, double lat)
{
    double sin_ha0, cos_ha0, sin_dec0, cos_dec0, sin_lat, cos_lat;
    int num_blocks, num_threads = 256;

    /* Set up thread blocks. */
    num_blocks = (num_points + num_threads - 1) / num_threads;

    /* Compute sines and cosines of Euler angles and call kernel to perform
     * the transformation. */
    sin_ha0  = sin(ha0);
    cos_ha0  = cos(ha0);
    sin_dec0 = sin(dec0);
    cos_dec0 = cos(dec0);
    sin_lat  = sin(lat);
    cos_lat  = cos(lat);
    oskar_convert_enu_directions_to_relative_directions_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (l, m, n, num_points, x, y, z,
            cos_ha0, sin_ha0, cos_dec0, sin_dec0, cos_lat, sin_lat);
}


/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_convert_enu_directions_to_relative_directions_cudak_f(
        float* __restrict__ l, float* __restrict__ m, float* __restrict__ n,
        const int num_points, const float* __restrict__ x,
        const float* __restrict__ y, const float* __restrict__ z,
        const float cos_ha0, const float sin_ha0, const float cos_dec0,
        const float sin_dec0, const float cos_lat, const float sin_lat)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_points) return;

    oskar_convert_enu_directions_to_relative_directions_inline_f(
            &l[i], &m[i], &n[i], x[i], y[i], z[i],
            cos_ha0, sin_ha0, cos_dec0, sin_dec0, cos_lat, sin_lat);
}

/* Double precision. */
__global__
void oskar_convert_enu_directions_to_relative_directions_cudak_d(
        double* __restrict__ l, double* __restrict__ m, double* __restrict__ n,
        const int num_points, const double* __restrict__ x,
        const double* __restrict__ y, const double* __restrict__ z,
        const double cos_ha0, const double sin_ha0, const double cos_dec0,
        const double sin_dec0, const double cos_lat, const double sin_lat)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_points) return;

    oskar_convert_enu_directions_to_relative_directions_inline_d(
            &l[i], &m[i], &n[i], x[i], y[i], z[i],
            cos_ha0, sin_ha0, cos_dec0, sin_dec0, cos_lat, sin_lat);
}

#ifdef __cplusplus
}
#endif
