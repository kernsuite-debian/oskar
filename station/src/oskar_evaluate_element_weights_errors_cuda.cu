#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-2015, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_evaluate_element_weights_errors_cuda.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_evaluate_element_weights_errors_cuda_f(int num_elements,
        const float* amp_gain, const float* amp_error,
        const float* phase_offset, const float* phase_error, float2* errors)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_elements + num_threads - 1) / num_threads;
    oskar_evaluate_element_weights_errors_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_elements,
            amp_gain, amp_error, phase_offset, phase_error, errors);
}

/* Double precision. */
void oskar_evaluate_element_weights_errors_cuda_d(int num_elements,
        const double* amp_gain, const double* amp_error,
        const double* phase_offset, const double* phase_error, double2* errors)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_elements + num_threads - 1) / num_threads;
    oskar_evaluate_element_weights_errors_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_elements,
            amp_gain, amp_error, phase_offset, phase_error, errors);
}


/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_evaluate_element_weights_errors_cudak_f(int num_elements,
        const float* restrict amp_gain, const float* restrict amp_error,
        const float* restrict phase_offset, const float* restrict phase_error,
        float2* errors)
{
    float2 r, t;

    /* Thread index is antenna element. */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_elements) return;

    /* Get two random numbers from a normalised Gaussian distribution. */
    r = errors[i];

    /* Evaluate the real and imaginary components of the error weight
     * for the antenna. */
    r.x *= amp_error[i];
    r.x += amp_gain[i]; /* Amplitude. */
    r.y *= phase_error[i];
    r.y += phase_offset[i]; /* Phase. */
    sincosf(r.y, &t.y, &t.x);
    t.x *= r.x; /* Real. */
    t.y *= r.x; /* Imaginary. */
    errors[i] = t; /* Store. */
}

/* Double precision. */
__global__
void oskar_evaluate_element_weights_errors_cudak_d(int num_elements,
        const double* restrict amp_gain, const double* restrict amp_error,
        const double* restrict phase_offset, const double* restrict phase_error,
        double2* errors)
{
    double2 r, t;

    /* Thread index is antenna element. */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_elements) return;

    /* Get two random numbers from a normalised Gaussian distribution. */
    r = errors[i];

    /* Evaluate the real and imaginary components of the error weight
     * for the antenna. */
    r.x *= amp_error[i];
    r.x += amp_gain[i]; /* Amplitude. */
    r.y *= phase_error[i];
    r.y += phase_offset[i]; /* Phase. */
    sincos(r.y, &t.y, &t.x);
    t.x *= r.x; /* Real. */
    t.y *= r.x; /* Imaginary. */
    errors[i] = t; /* Store. */
}

#ifdef __cplusplus
}
#endif
