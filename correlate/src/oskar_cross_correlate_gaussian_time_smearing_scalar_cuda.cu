#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014-2015, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_cross_correlate_gaussian_time_smearing_scalar_cuda.h>
#include <private_correlate_functions_inline.h>

/* Kernels. ================================================================ */

/* Indices into the visibility/baseline matrix. */
#define SP blockIdx.x /* Column index. */
#define SQ blockIdx.y /* Row index. */

extern __shared__ float2  smem_f[];
extern __shared__ double2 smem_d[];

/* Single precision. */
__global__
void oskar_cross_correlate_gaussian_time_smearing_scalar_cudak_f(
        const int num_sources, const int num_stations,
        const float2* restrict jones, const float* restrict source_I,
        const float* restrict source_l, const float* restrict source_m,
        const float* restrict source_n, const float* restrict source_a,
        const float* restrict source_b, const float* restrict source_c,
        const float* restrict station_u, const float* restrict station_v,
        const float* restrict station_w, const float* restrict station_x,
        const float* restrict station_y, const float uv_min_lambda,
        const float uv_max_lambda, const float inv_wavelength,
        const float frac_bandwidth, const float time_int_sec,
        const float gha0_rad, const float dec0_rad, float2* restrict vis)
{
    __shared__ float uv_len, uu, vv, ww, uu2, vv2, uuvv, du_dt, dv_dt, dw_dt;
    float2 sum;
    float l, m, n, r1, r2;
    int i;

    /* Return immediately if in the wrong half of the visibility matrix. */
    if (SQ >= SP) return;

    /* Get common baseline values per thread block. */
    if (threadIdx.x == 0)
    {
        oskar_evaluate_baseline_terms_inline_f(station_u[SP],
                station_u[SQ], station_v[SP], station_v[SQ],
                station_w[SP], station_w[SQ], inv_wavelength,
                frac_bandwidth, &uv_len, &uu, &vv, &ww, &uu2, &vv2, &uuvv);

        /* Compute the derivatives for time-average smearing. */
        oskar_evaluate_baseline_derivatives_inline_f(station_x[SP],
                station_x[SQ], station_y[SP], station_y[SQ], inv_wavelength,
                time_int_sec, gha0_rad, dec0_rad, &du_dt, &dv_dt, &dw_dt);
    }
    __syncthreads();

    /* Apply the baseline length filter. */
    if (uv_len < uv_min_lambda || uv_len > uv_max_lambda)
        return;

    /* Get pointers to source vectors for both stations. */
    const float2* restrict station_p = &jones[num_sources * SP];
    const float2* restrict station_q = &jones[num_sources * SQ];

    /* Each thread loops over a subset of the sources. */
    sum = make_float2(0.0f, 0.0f); /* Partial sum per thread. */
    for (i = threadIdx.x; i < num_sources; i += blockDim.x)
    {
        /* Get source direction cosines. */
        l = source_l[i];
        m = source_m[i];
        n = source_n[i];

        /* Compute bandwidth- and time-smearing terms. */
        r1 = oskar_sinc_f(uu * l + vv * m + ww * (n - 1.0f));
        r2 = oskar_evaluate_time_smearing_f(du_dt, dv_dt, dw_dt, l, m, n);
        r1 *= r2;

        /* Evaluate Gaussian source width term. */
        r2 = expf(-(source_a[i] * uu2 + source_b[i] * uuvv +
                source_c[i] * vv2));
        r1 *= r2;

        /* Accumulate baseline visibility response for source. */
        oskar_accumulate_baseline_visibility_for_source_scalar_inline_f(
                &sum, i, source_I, station_p, station_q, r1);
    }

    /* Store partial sum for the thread in shared memory and synchronise. */
    smem_f[threadIdx.x] = sum;
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        for (i = 1; i < blockDim.x; ++i)
        {
            sum.x += smem_f[i].x;
            sum.y += smem_f[i].y;
        }

        /* Add result of this thread block to the baseline visibility. */
        i = oskar_evaluate_baseline_index_inline(num_stations, SP, SQ);
        vis[i].x += sum.x;
        vis[i].y += sum.y;
    }
}

/* Double precision. */
__global__
void oskar_cross_correlate_gaussian_time_smearing_scalar_cudak_d(
        const int num_sources, const int num_stations,
        const double2* restrict jones, const double* restrict source_I,
        const double* restrict source_l, const double* restrict source_m,
        const double* restrict source_n, const double* restrict source_a,
        const double* restrict source_b, const double* restrict source_c,
        const double* restrict station_u, const double* restrict station_v,
        const double* restrict station_w, const double* restrict station_x,
        const double* restrict station_y, const double uv_min_lambda,
        const double uv_max_lambda, const double inv_wavelength,
        const double frac_bandwidth, const double time_int_sec,
        const double gha0_rad, const double dec0_rad, double2* restrict vis)
{
    __shared__ double uv_len, uu, vv, ww, uu2, vv2, uuvv, du_dt, dv_dt, dw_dt;
    double2 sum;
    double l, m, n, r1, r2;
    int i;

    /* Return immediately if in the wrong half of the visibility matrix. */
    if (SQ >= SP) return;

    /* Get common baseline values per thread block. */
    if (threadIdx.x == 0)
    {
        oskar_evaluate_baseline_terms_inline_d(station_u[SP],
                station_u[SQ], station_v[SP], station_v[SQ],
                station_w[SP], station_w[SQ], inv_wavelength,
                frac_bandwidth, &uv_len, &uu, &vv, &ww, &uu2, &vv2, &uuvv);

        /* Compute the derivatives for time-average smearing. */
        oskar_evaluate_baseline_derivatives_inline_d(station_x[SP],
                station_x[SQ], station_y[SP], station_y[SQ], inv_wavelength,
                time_int_sec, gha0_rad, dec0_rad, &du_dt, &dv_dt, &dw_dt);
    }
    __syncthreads();

    /* Apply the baseline length filter. */
    if (uv_len < uv_min_lambda || uv_len > uv_max_lambda)
        return;

    /* Get pointers to source vectors for both stations. */
    const double2* restrict station_p = &jones[num_sources * SP];
    const double2* restrict station_q = &jones[num_sources * SQ];

    /* Each thread loops over a subset of the sources. */
    sum = make_double2(0.0, 0.0); /* Partial sum per thread. */
    for (i = threadIdx.x; i < num_sources; i += blockDim.x)
    {
        /* Get source direction cosines. */
        l = source_l[i];
        m = source_m[i];
        n = source_n[i];

        /* Compute bandwidth-smearing term. */
        r1 = oskar_sinc_d(uu * l + vv * m + ww * (n - 1.0));
        r2 = oskar_evaluate_time_smearing_d(du_dt, dv_dt, dw_dt, l, m, n);
        r1 *= r2;

        /* Evaluate Gaussian source width term. */
        r2 = exp(-(source_a[i] * uu2 + source_b[i] * uuvv +
                source_c[i] * vv2));
        r1 *= r2;

        /* Accumulate baseline visibility response for source. */
        oskar_accumulate_baseline_visibility_for_source_scalar_inline_d(
                &sum, i, source_I, station_p, station_q, r1);
    }

    /* Store partial sum for the thread in shared memory and synchronise. */
    smem_d[threadIdx.x] = sum;
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        for (i = 1; i < blockDim.x; ++i)
        {
            sum.x += smem_d[i].x;
            sum.y += smem_d[i].y;
        }

        /* Add result of this thread block to the baseline visibility. */
        i = oskar_evaluate_baseline_index_inline(num_stations, SP, SQ);
        vis[i].x += sum.x;
        vis[i].y += sum.y;
    }
}

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_cross_correlate_gaussian_time_smearing_scalar_cuda_f(
        int num_sources, int num_stations, const float2* d_jones,
        const float* d_source_I, const float* d_source_l,
        const float* d_source_m, const float* d_source_n,
        const float* d_source_a, const float* d_source_b,
        const float* d_source_c, const float* d_station_u,
        const float* d_station_v, const float* d_station_w,
        const float* d_station_x, const float* d_station_y,
        float uv_min_lambda, float uv_max_lambda, float inv_wavelength,
        float frac_bandwidth, float time_int_sec, float gha0_rad,
        float dec0_rad, float2* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(float2);
    oskar_cross_correlate_gaussian_time_smearing_scalar_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_l, d_source_m,
            d_source_n, d_source_a, d_source_b, d_source_c, d_station_u,
            d_station_v, d_station_w, d_station_x, d_station_y, uv_min_lambda,
            uv_max_lambda, inv_wavelength, frac_bandwidth, time_int_sec,
            gha0_rad, dec0_rad, d_vis);
}

/* Double precision. */
void oskar_cross_correlate_gaussian_time_smearing_scalar_cuda_d(
        int num_sources, int num_stations, const double2* d_jones,
        const double* d_source_I, const double* d_source_l,
        const double* d_source_m, const double* d_source_n,
        const double* d_source_a, const double* d_source_b,
        const double* d_source_c, const double* d_station_u,
        const double* d_station_v, const double* d_station_w,
        const double* d_station_x, const double* d_station_y,
        double uv_min_lambda, double uv_max_lambda, double inv_wavelength,
        double frac_bandwidth, double time_int_sec, double gha0_rad,
        double dec0_rad, double2* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(double2);
    oskar_cross_correlate_gaussian_time_smearing_scalar_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_l, d_source_m,
            d_source_n, d_source_a, d_source_b, d_source_c, d_station_u,
            d_station_v, d_station_w, d_station_x, d_station_y, uv_min_lambda,
            uv_max_lambda, inv_wavelength, frac_bandwidth, time_int_sec,
            gha0_rad, dec0_rad, d_vis);
}

#ifdef __cplusplus
}
#endif
